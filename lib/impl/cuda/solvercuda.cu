#include "hip/hip_runtime.h"
/** @addtogroup Solver
 * @{*/

/** @file */


#include <valarray>
#include "solvercuda.hpp"

#include <iostream>

namespace NNSimulator {

    __device__ void solveExplicitEulerKernelD(  
        const float *dt, 
        const float *I, 
        const float *VPeak, 
        const float *VReset, 
        const float *paramSpec, 
        float *t, 
        float *V,
        bool *mask 
    ) 
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if( mask[i] )
        {
            V[i] = *VReset;
        } else {
            V[i] += (*paramSpec)*I[i]*(*dt);
        }
        *t += *dt;
        mask[i] = V[i] > *VPeak;
    }

    __global__ void solveExplicitEulerKernelG( 
        const float *dt, 
        const float *I, 
        const float *VPeak, 
        const float *VReset, 
        const float *paramSpec, 
        float *t, 
        float *V,
        bool *mask 
    ) 
    {
        solveExplicitEulerKernelD( dt, I, VPeak, VReset, paramSpec, t, V, mask );
    }

    //! Полная специализация метода solveExplicitEuler для float.
    template<> 
    void SolverImplCuda<float>::solveExplicitEuler(
        std::unique_ptr<NNSimulator::Neurs<float>> & neurs,
        std::unique_ptr<NNSimulator::Conns<float>> & conns,
        const float & dt,
        const float & simulationTime 
    ) 
    {
        // tmp

        float t = 0;
        float VPeak = 1;
        float VReset = 1;
        float paramSpec = 1;
        std::valarray<float> V = {1,0,1};
        std::valarray<float> I = {1,0,1};
        std::valarray<bool> mask = {1,0,1};
        // end tmp

        size_t n = I.size();
        size_t nSize = n*sizeof(float);
        size_t bSize = n*sizeof(bool);
        size_t size = sizeof(float);

        float *dtDev;
        float *tDev;
        float *paramSpecDev;
        float *VPeakDev;
        float *VResetDev;
        float *IDev;
        float *VDev;
        bool *maskDev;

        hipMalloc( (void**)&dtDev, size);
        hipMalloc( (void**)&tDev, size);
        hipMalloc( (void**)&paramSpecDev, size);
        hipMalloc( (void**)&VPeakDev, size);
        hipMalloc( (void**)&VResetDev, size);
        hipMalloc( (void**)&IDev, nSize);
        hipMalloc( (void**)&VDev, nSize);
        hipMalloc( (void**)&maskDev, bSize);

        hipMemcpy( dtDev, &dt, size, hipMemcpyHostToDevice );
        hipMemcpy( tDev, &t, size, hipMemcpyHostToDevice );
        hipMemcpy( paramSpecDev, &paramSpec, size, hipMemcpyHostToDevice );
        hipMemcpy( VPeakDev, &VPeak, size, hipMemcpyHostToDevice );
        hipMemcpy( VResetDev, &VReset, size, hipMemcpyHostToDevice );
        hipMemcpy( IDev, &I[0], nSize, hipMemcpyHostToDevice );
        hipMemcpy( VDev, &V[0], nSize, hipMemcpyHostToDevice );
        hipMemcpy( maskDev, &mask[0], bSize, hipMemcpyHostToDevice );

        solveExplicitEulerKernelG<<< 1, n >>>( dtDev, IDev, VPeakDev, VResetDev, paramSpecDev, tDev, VDev, maskDev );

        hipMemcpy( &t, tDev, size, hipMemcpyDeviceToHost );
        hipMemcpy( &V[0], VDev, nSize, hipMemcpyDeviceToHost );
        hipMemcpy( &mask[0], maskDev, bSize, hipMemcpyDeviceToHost );

        hipFree( dtDev );
        hipFree( IDev );
        hipFree( VPeakDev );
        hipFree( VResetDev );
        hipFree( paramSpecDev );
        hipFree( tDev );
        hipFree( VDev );
        hipFree( maskDev );

    }

}

