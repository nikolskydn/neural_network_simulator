#include "hip/hip_runtime.h"
#include <valarray>
#include <cmath>
#include "solvercuda.hpp"
#include <hiprand/hiprand_kernel.h>

#include <iostream>

#define CudaTimeDebug 1

namespace NNSimulator {

    __global__ void solvePCNNI2003KernelG( 
        const int *nN,
        const int *nE,
        const float *VP,
        const float *a,
        const float *b,
        const float *c,
        const float *d,
        const float *dt,
        const float *te,
        float *V, 
        float *U, 
        bool *m, 
        float *I, 
        float *w, 
        float *t
        //, 
        //float *spk, 
        //float *osc 
    ) 
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        hiprandState s;
        hiprand_init(i, 0, 0, &s);
        int ct = *t;
        while( ct< (*te) )
        {
            if( m[i] ) 
            {
                V[i] = c[i];
                U[i] += d[i];
            }
            if( i<(*nE) ) 
                I[i] = 5.*hiprand_uniform(&s); 
            else 
                I[i] = 2.*hiprand_uniform(&s);
            for( size_t j=0; j<(*nN); ++j ) if( m[j] ) I[i] += w[i*(*nN)+j];
            V[i] += .5*(*dt)*( .04*V[i]*V[i] + 5.*V[i] + 140. - U[i] + I[i] );
            V[i] += .5*(*dt)*( .04*V[i]*V[i] + 5.*V[i] + 140. - U[i] + I[i] );
            U[i] += (*dt)*a[i]*( b[i]*V[i] - U[i] );
            m[i] = V[i] > (*VP);
            ct += *dt;
        }
        if( i==0 ) *t = ct;
    }

    //! Полная специализация метода solvePCNNI2003E для float.
    template<> 
    void SolverImplCuda<float>::solvePCNNI2003E
    (
        const size_t & nN,
        const size_t & nE,
        const float & VP,
        const float & VR,
        const std::valarray<float> aN,
        const std::valarray<float> bN,
        const std::valarray<float> cN,
        const std::valarray<float> dN,
        const float &  dt,
        const float & te,
        std::valarray<float> & VN,
        std::valarray<float> & UN,
        std::valarray<bool> & mN,
        std::valarray<float> & IN,
        std::valarray<float> & wC,
        float & t,
        std::vector<std::pair<size_t,float>> & spk,
        std::vector<std::pair<size_t,std::valarray<float>>> & osc
    ) 
    {

        size_t fnN = nN*sizeof(float);
        size_t nC = nN*nN;
        size_t fnC = nC*sizeof(float);
        size_t bnN = nN*sizeof(bool);
        size_t f1 = sizeof(float);
        size_t i1 = sizeof(int);
        //size_t fnS = 100000*sizeof(float);
        //size_t fnO = 100000*sizeof(float);

        int *nND;   hipMalloc( (void**)&nND, i1 );
        int *nED;   hipMalloc( (void**)&nED, i1 );
        float *VPD;   hipMalloc( (void**)&VPD, f1 );
        //float *VRD;   hipMalloc( (void**)&VRD, f1 );
        float *aND;   hipMalloc( (void**)&aND, fnN );
        float *bND;   hipMalloc( (void**)&bND, fnN );
        float *cND;   hipMalloc( (void**)&cND, fnN );
        float *dND;   hipMalloc( (void**)&dND, fnN );
        float *dtD;   hipMalloc( (void**)&dtD, f1 );
        float *teD;   hipMalloc( (void**)&teD, f1 );
        float *VND;   hipMalloc( (void**)&VND, fnN );
        float *UND;   hipMalloc( (void**)&UND, fnN );
        bool *mND;    hipMalloc( (void**)&mND, bnN );
        float *IND;   hipMalloc( (void**)&IND, fnN );
        float *wCD;   hipMalloc( (void**)&wCD, fnC );
        float *tD;    hipMalloc( (void**)&tD, f1 );
        //float *spkD;  hipMalloc( (void**)&spkD, fnS );
        //float *oscD;  hipMalloc( (void**)&oscD, fnO );

        int tmpnN = static_cast<int>(nN);
        int tmpnE = static_cast<int>(nE);
        hipMemcpy( nND,  &tmpnN,     i1,   hipMemcpyHostToDevice );
        hipMemcpy( nED,  &tmpnE,     i1,   hipMemcpyHostToDevice );
        hipMemcpy( VPD,  &VP,     f1,   hipMemcpyHostToDevice );
        //hipMemcpy( VRD,  &VR,     size,   hipMemcpyHostToDevice );
        hipMemcpy( aND,  &aN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( bND,  &bN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( cND,  &cN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( dND,  &dN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( dtD,  &dt,     f1,   hipMemcpyHostToDevice );
        hipMemcpy( teD,  &te,     f1,   hipMemcpyHostToDevice );
        hipMemcpy( VND,  &VN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( UND,  &UN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( mND,  &mN[0],  bnN,  hipMemcpyHostToDevice );
        hipMemcpy( IND,  &IN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( wCD,  &wC[0],  fnC,  hipMemcpyHostToDevice );
        hipMemcpy( tD,   &t,      f1,   hipMemcpyHostToDevice );
        //hipMemcpy( spkD, &spk[0], fnS,  hipMemcpyHostToDevice );
        //hipMemcpy( oscD, &osc[0], fnO,  hipMemcpyHostToDevice );

#if CudaTimeDebug>0
        hipEvent_t bTime, eTime;
        float cudaTime = .0f;
        hipEventCreate( &bTime );
        hipEventCreate( &eTime );
        hipEventRecord( bTime, 0 );
#endif
        solvePCNNI2003KernelG<<< 1, nN >>>
        ( 
            nND, nED, VPD, aND, bND, cND, dND, dtD, teD, 
            VND, UND, mND, IND, wCD, tD /*, spkD, oscD */
        );

#if CudaTimeDebug>0
        hipEventRecord( eTime, 0 );
        hipEventSynchronize( eTime );
        hipEventElapsedTime( &cudaTime, bTime, eTime );
        std::cout << "Neurons: " << nN << std::endl;
        std::cout << "Connects: " << nN*nN << std::endl;
        std::cout << "Steps: " << std::ceil(te/dt) << std::endl;
        std::cout << "solvePCNNI2003KernelG time: " << cudaTime << ", ms\n\n";
#endif

        hipMemcpy( &t,      tD,   f1,   hipMemcpyDeviceToHost );
        hipMemcpy( &VN[0],  VND,  fnN,  hipMemcpyDeviceToHost );
        hipMemcpy( &mN[0],  mND,  bnN,  hipMemcpyDeviceToHost );
        hipMemcpy( &IN[0],  IND,  fnN,  hipMemcpyDeviceToHost );


        hipFree( nND );
        hipFree( nED );
        hipFree( VPD );
        //hipFree( VRD );
        hipFree( aND );
        hipFree( bND );
        hipFree( cND );
        hipFree( dND );
        hipFree( dtD );
        hipFree( teD );
        hipFree( VND );
        hipFree( UND );
        hipFree( mND );
        hipFree( IND );
        hipFree( wCD );
        hipFree( tD );
        //hipFree( spkD );
        //hipFree( oscD );

    }

}

