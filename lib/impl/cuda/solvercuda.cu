#include "hip/hip_runtime.h"
#include <valarray>
#include <vector>
#include <cmath>
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "solvercuda.hpp"
#include "../../setting.h"

namespace NNSimulator {

    __global__ void solvePCNNI2003KernelG( 
        const size_t *nN,
        const size_t *nE,
        const float *VP,
        const float *a,
        const float *b,
        const float *c,
        const float *d,
        const float *w, 
        const float *dt,
        const float *te,
        float *V, 
        float *U, 
        bool *m, 
        float *I, 
        float *t,
        float *osc 
    ) 
    {
        size_t i = threadIdx.x + blockIdx.x * blockDim.x;
        hiprandState state;
        hiprand_init(i, 0, 0, &state);
        float ct = *t;
        int cn = 0; 
        while( ct< (*te) )
        {
            if( m[i] ) 
            {
                V[i] = c[i];
                U[i] += d[i];
            }
            #ifndef NN_TEST_SOLVERS
                if( i<(*nE) ) 
                    I[i] = 5.*hiprand_normal(&state); 
                else 
                    I[i] = 2.*hiprand_normal(&state);
            #endif
            for( size_t j=0; j<(*nN); ++j ) if( m[j] ) I[i] += w[i*(*nN)+j];
            V[i] += .5*(*dt)*( .04*V[i]*V[i] + 5.*V[i] + 140. - U[i] + I[i] );
            V[i] += .5*(*dt)*( .04*V[i]*V[i] + 5.*V[i] + 140. - U[i] + I[i] );
            U[i] += (*dt)*a[i]*( b[i]*V[i] - U[i] );
            m[i] = V[i] > (*VP);
            ct += *dt;

            if( i==0  ) osc[ cn*(*nN+1) ] = ct;
            osc[ cn*(*nN+1)+i+1 ] = V[i] ;
            ++cn;

        }
        if( i==0 ) *t = ct;
    }

    //! Полная специализация метода solvePCNNI2003E для float.
    template<> 
    void SolverImplCuda<float>::solvePCNNI2003E
    (
        const size_t & nN,
        const size_t & nE,
        const float & VP,
        const std::valarray<float> aN,
        const std::valarray<float> bN,
        const std::valarray<float> cN,
        const std::valarray<float> dN,
        const std::valarray<float> & wC,
        const float &  dt,
        const float & te,
        std::valarray<float> & VN,
        std::valarray<float> & UN,
        std::valarray<bool> & mN,
        std::valarray<float> & IN,
        float & t,
        std::deque<std::pair<float,std::valarray<float>>> & og
    ) 
    {

        size_t fnN = nN*sizeof(float);
        size_t nC = nN*nN;
        size_t fnC = nC*sizeof(float);
        size_t bnN = nN*sizeof(bool);
        size_t f1 = sizeof(float);
        size_t s1 = sizeof(size_t);
        size_t nSteps = std::ceil((te-t)/dt);
        size_t fnO = (nN+1)*nSteps*sizeof(float);


        size_t *nND;  hipMalloc( (void**)&nND, s1 );
        size_t *nED;  hipMalloc( (void**)&nED, s1 );
        float *VPD;   hipMalloc( (void**)&VPD, f1 );
        float *aND;   hipMalloc( (void**)&aND, fnN );
        float *bND;   hipMalloc( (void**)&bND, fnN );
        float *cND;   hipMalloc( (void**)&cND, fnN );
        float *dND;   hipMalloc( (void**)&dND, fnN );
        float *dtD;   hipMalloc( (void**)&dtD, f1 );
        float *teD;   hipMalloc( (void**)&teD, f1 );
        float *VND;   hipMalloc( (void**)&VND, fnN );
        float *UND;   hipMalloc( (void**)&UND, fnN );
        bool *mND;    hipMalloc( (void**)&mND, bnN );
        float *IND;   hipMalloc( (void**)&IND, fnN );
        float *wCD;   hipMalloc( (void**)&wCD, fnC );
        float *tD;    hipMalloc( (void**)&tD, f1 );
        float *oscD;  hipMalloc( (void**)&oscD, fnO );

        hipMemcpy( nND,  &nN,     s1,   hipMemcpyHostToDevice );
        hipMemcpy( nED,  &nE,     s1,   hipMemcpyHostToDevice );
        hipMemcpy( VPD,  &VP,     f1,   hipMemcpyHostToDevice );
        hipMemcpy( aND,  &aN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( bND,  &bN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( cND,  &cN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( dND,  &dN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( dtD,  &dt,     f1,   hipMemcpyHostToDevice );
        hipMemcpy( teD,  &te,     f1,   hipMemcpyHostToDevice );
        hipMemcpy( VND,  &VN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( UND,  &UN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( mND,  &mN[0],  bnN,  hipMemcpyHostToDevice );
        hipMemcpy( IND,  &IN[0],  fnN,  hipMemcpyHostToDevice );
        hipMemcpy( wCD,  &wC[0],  fnC,  hipMemcpyHostToDevice );
        hipMemcpy( tD,   &t,      f1,   hipMemcpyHostToDevice );

        #ifdef TimeDebug
            hipEvent_t bTime, eTime;
            float cudaTime = .0f;
            hipEventCreate( &bTime );
            hipEventCreate( &eTime );
            hipEventRecord( bTime, 0 );
        #endif
        solvePCNNI2003KernelG<<< 1, nN >>>
        ( 
            nND, nED, VPD, aND, bND, cND, dND, wCD,  dtD, teD, 
            VND, UND, mND, IND, tD, oscD
        );

        #ifdef TimeDebug
            hipEventRecord( eTime, 0 );
            hipEventSynchronize( eTime );
            hipEventElapsedTime( &cudaTime, bTime, eTime );
            std::cout << "Neurons: " << nN << std::endl;
            std::cout << "Connects: " << nN*nN << std::endl;
            std::cout << "Steps: " << nSteps << std::endl;
            std::cout << "solvePCNNI2003KernelG time: " << cudaTime << ", ms\n\n";
        #endif

        hipMemcpy( &t,      tD,   f1,   hipMemcpyDeviceToHost );
        hipMemcpy( &VN[0],  VND,  fnN,  hipMemcpyDeviceToHost );
        hipMemcpy( &UN[0],  UND,  fnN,  hipMemcpyDeviceToHost );
        hipMemcpy( &mN[0],  mND,  bnN,  hipMemcpyDeviceToHost );
        hipMemcpy( &IN[0],  IND,  fnN,  hipMemcpyDeviceToHost );

        float tTmp;
        std::valarray<float> VTmp(nN);
        for(int i=0; i<nSteps; ++i)
        {
            hipMemcpy( &tTmp,  &oscD[ i*(nN+1) ], f1,  hipMemcpyDeviceToHost );
            hipMemcpy( &VTmp[0],  &oscD[ i*(nN+1)+1 ], fnN,  hipMemcpyDeviceToHost );
            og.push_back(std::pair<float,std::valarray<float>>(tTmp,VTmp));
        }

        hipFree( nND );
        hipFree( nED );
        hipFree( VPD );
        hipFree( aND );
        hipFree( bND );
        hipFree( cND );
        hipFree( dND );
        hipFree( dtD );
        hipFree( teD );
        hipFree( VND );
        hipFree( UND );
        hipFree( mND );
        hipFree( IND );
        hipFree( wCD );
        hipFree( tD );
        hipFree( oscD );

    }

}

