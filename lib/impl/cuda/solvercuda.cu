#include "hip/hip_runtime.h"
/** @addtogroup Solver
 * @{*/

/** @file */

#include <valarray>
#include "solvercuda.hpp"

#include <iostream>

namespace NNSimulator {

    __global__ void solveTestKernelG(  
        const float *VP,
        const float *VR,
        const float *dt,
        const float *st,
        const float *np,
        const float *cp,
        float *V,
        bool *m,
        float *I,
        float *w,
        float *t
    ) 
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if( m[i] ) V[i] = *VR;
        else V[i] += (*np)*I[i]*(*dt);
        m[i] = V[i] > *VP;
        if(V[i]>(*VP)) I[i] += (*cp)*V[i]*(*dt);
        else I[i] *= 0.5;
        *t += *dt;

    }

    //! Полная специализация метода solveTest для float.
    template<> 
    void SolverImplCuda<float>::solveTest(
        const size_t & nN, // nNeurs
        const float & VP, // VPeak
        const float & VR, // VReset
        const float &  dt,
        const float & st, // simulationTime
        const float & np, // neuronsParamSpec
        const float & cp, // connectsParamSpec
        std::valarray<float> & V,
        std::valarray<bool> & m, // mask
        std::valarray<float> & I,
        std::valarray<float> & w, // weight
        float & t
    ) 
    {
        size_t nSize = nN*sizeof(float);
        size_t bSize = nN*sizeof(bool);
        size_t size = sizeof(float);

        float *VPD;
        float *VRD;
        float *dtD;
        float *stD;
        float *npD;
        float *cpD;
        float *VD;  
        bool *mD;  
        float *ID;  
        float *wD; 
        float *tD;

        hipMalloc( (void**)&VPD, size);
        hipMalloc( (void**)&VRD, size);
        hipMalloc( (void**)&dtD, size);
        hipMalloc( (void**)&stD, size);
        hipMalloc( (void**)&npD, size);
        hipMalloc( (void**)&cpD, size);
        hipMalloc( (void**)&VD, nSize);
        hipMalloc( (void**)&mD, bSize);
        hipMalloc( (void**)&ID, nSize);
        hipMalloc( (void**)&wD, nSize*nSize);
        hipMalloc( (void**)&tD, size);

        hipMemcpy( VPD, &VP, size, hipMemcpyHostToDevice );
        hipMemcpy( VRD, &VR, size, hipMemcpyHostToDevice );
        hipMemcpy( dtD, &dt, size, hipMemcpyHostToDevice );
        hipMemcpy( stD, &st, size, hipMemcpyHostToDevice );
        hipMemcpy( npD, &np, size, hipMemcpyHostToDevice );
        hipMemcpy( cpD, &cp, size, hipMemcpyHostToDevice );
        hipMemcpy( VD, &V[0], nSize, hipMemcpyHostToDevice );
        hipMemcpy( mD, &m[0], bSize, hipMemcpyHostToDevice );
        hipMemcpy( ID, &I[0], nSize, hipMemcpyHostToDevice );
        hipMemcpy( wD, &w[0], nSize*nSize, hipMemcpyHostToDevice );
        hipMemcpy( tD, &t, size, hipMemcpyHostToDevice );

        solveTestKernelG<<< 1, nN >>>( VPD, VRD, dtD, stD, npD, cpD, VD, mD,  ID,  wD, tD);

        hipMemcpy( &t, tD, size, hipMemcpyDeviceToHost );
        hipMemcpy( &V[0], VD, nSize, hipMemcpyDeviceToHost );
        hipMemcpy( &m[0], mD, bSize, hipMemcpyDeviceToHost );
        hipMemcpy( &I[0], ID, nSize, hipMemcpyDeviceToHost );

        hipFree( VPD );
        hipFree( VRD );
        hipFree( dtD );
        hipFree( stD );
        hipFree( npD );
        hipFree( cpD );
        hipFree( VD );
        hipFree( mD );
        hipFree( ID );
        hipFree( wD );
        hipFree( tD );

    }

}

