#include "hip/hip_runtime.h"
/** @addtogroup Conns
 * @{*/

/** @file */


#include <valarray>
#include "connscuda.hpp"

#include <iostream>

namespace NNSimulator {

    __global__ void performStepTimeSpecKernel( const float *dt, const float *paramSpec, const float *V, float *t, float *I ) 
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if(V[i]>10) I[i] += (*paramSpec)*V[i]*(*dt);
        else I[i] *= 0.5;
        *t += *dt;
    }


    //! Полная специализация метода performStepTimeSpec() для float.
    template<> 
    void ConnsImplCuda<float>::performStepTimeSpec(
        const float & dt, 
        const float & paramSpec,
        const std::valarray<float> & V,
        float & t,
        std::valarray<float> & I
    ) 
    {
        size_t n = I.size();
        size_t nSize = n*sizeof(float);
        size_t size = sizeof(float);

        float *dtDev;
        float *tDev;
        float *paramSpecDev;
        float *IDev;
        float *VDev;

        hipMalloc( (void**)&dtDev, size);
        hipMalloc( (void**)&tDev, size);
        hipMalloc( (void**)&paramSpecDev, size);
        hipMalloc( (void**)&IDev, nSize);
        hipMalloc( (void**)&VDev, nSize);

        hipMemcpy( dtDev, &dt, size, hipMemcpyHostToDevice );
        hipMemcpy( tDev, &t, size, hipMemcpyHostToDevice );
        hipMemcpy( paramSpecDev, &paramSpec, size, hipMemcpyHostToDevice );
        hipMemcpy( IDev, &I[0], nSize, hipMemcpyHostToDevice );
        hipMemcpy( VDev, &V[0], nSize, hipMemcpyHostToDevice );

        performStepTimeSpecKernel<<< 1, n >>>( dtDev, paramSpecDev, VDev, tDev, IDev );

        hipMemcpy( &t, tDev, size, hipMemcpyDeviceToHost );
        hipMemcpy( &I[0], IDev, nSize, hipMemcpyDeviceToHost );

        hipFree( dtDev );
        hipFree( paramSpecDev );
        hipFree( IDev );
        hipFree( tDev );
        hipFree( VDev );

    }

}


/*@}*/
