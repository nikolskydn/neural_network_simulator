#include "hip/hip_runtime.h"
/** @addtogroup Neurs
 * @{*/

/** @file */


#include <valarray>
#include "neurscuda.hpp"

#include <iostream>

namespace NNSimulator {

    __global__ void performStepTimeSpecKernel( const float *dt, const float *paramSpec, const float *I, float *t, float *V ) 
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        V[i] += (*paramSpec)*I[i]*(*dt);
        *t += *dt;
    }


    //! Полная специализация метода performStepTimeSpec() для float.
    template<> 
    void NeursImplCuda<float>::performStepTimeSpec(
        const float & dt, 
        const float & paramSpec,
        const std::valarray<float> & I,
        float & t,
        std::valarray<float> & V 
    ) 
    {
        size_t n = I.size();
        size_t nSize = n*sizeof(float);
        size_t size = sizeof(float);

        float *dtDev;
        float *tDev;
        float *paramSpecDev;
        float *IDev;
        float *VDev;

        hipMalloc( (void**)&dtDev, size);
        hipMalloc( (void**)&tDev, size);
        hipMalloc( (void**)&paramSpecDev, size);
        hipMalloc( (void**)&IDev, nSize);
        hipMalloc( (void**)&VDev, nSize);

        hipMemcpy( dtDev, &dt, size, hipMemcpyHostToDevice );
        hipMemcpy( tDev, &t, size, hipMemcpyHostToDevice );
        hipMemcpy( paramSpecDev, &paramSpec, size, hipMemcpyHostToDevice );
        hipMemcpy( IDev, &I[0], nSize, hipMemcpyHostToDevice );
        hipMemcpy( VDev, &V[0], nSize, hipMemcpyHostToDevice );

        performStepTimeSpecKernel<<< 1, n >>>( dtDev, paramSpecDev, IDev, tDev, VDev );

        hipMemcpy( &t, tDev, size, hipMemcpyDeviceToHost );
        hipMemcpy( &V[0], VDev, nSize, hipMemcpyDeviceToHost );

        hipFree( dtDev );
        hipFree( paramSpecDev );
        hipFree( IDev );
        hipFree( tDev );
        hipFree( VDev );

    }

}

