#include "hip/hip_runtime.h"
/** @addtogroup Neurs
 * @{*/

/** @file */


#include <valarray>
#include "neurscuda.hpp"

#include <iostream>

namespace NNSimulator {

    __global__ void performStepTimeSpecKernel( 
        const float *dt, 
        const float *I, 
        const float *VPeak, 
        const float *VReset, 
        const float *paramSpec, 
        float *t, 
        float *V,
        bool *mask 
    ) 
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if( mask[i] )
        {
            V[i] = *VReset;
        } else {
            V[i] += (*paramSpec)*I[i]*(*dt);
        }
        *t += *dt;
        mask[i] = V[i] > *VPeak;
    }


    //! Полная специализация метода performStepTimeSpec() для float.
    template<> 
    void NeursImplCuda<float>::performStepTimeSpec(
        const float & dt, 
        const std::valarray<float> & I,
        const float & VPeak,
        const float & VReset,
        const float & paramSpec,
        float & t,
        std::valarray<float> & V, 
        std::valarray<bool> & mask
    ) 
    {
        size_t n = I.size();
        size_t nSize = n*sizeof(float);
        size_t bSize = n*sizeof(bool);
        size_t size = sizeof(float);

        float *dtDev;
        float *tDev;
        float *paramSpecDev;
        float *VPeakDev;
        float *VResetDev;
        float *IDev;
        float *VDev;
        bool *maskDev;

        hipMalloc( (void**)&dtDev, size);
        hipMalloc( (void**)&tDev, size);
        hipMalloc( (void**)&paramSpecDev, size);
        hipMalloc( (void**)&VPeakDev, size);
        hipMalloc( (void**)&VResetDev, size);
        hipMalloc( (void**)&IDev, nSize);
        hipMalloc( (void**)&VDev, nSize);
        hipMalloc( (void**)&maskDev, bSize);

        hipMemcpy( dtDev, &dt, size, hipMemcpyHostToDevice );
        hipMemcpy( tDev, &t, size, hipMemcpyHostToDevice );
        hipMemcpy( paramSpecDev, &paramSpec, size, hipMemcpyHostToDevice );
        hipMemcpy( VPeakDev, &VPeak, size, hipMemcpyHostToDevice );
        hipMemcpy( VResetDev, &VReset, size, hipMemcpyHostToDevice );
        hipMemcpy( IDev, &I[0], nSize, hipMemcpyHostToDevice );
        hipMemcpy( VDev, &V[0], nSize, hipMemcpyHostToDevice );
        hipMemcpy( maskDev, &mask[0], bSize, hipMemcpyHostToDevice );

        performStepTimeSpecKernel<<< 1, n >>>( dtDev, IDev, VPeakDev, VResetDev, paramSpecDev, tDev, VDev, maskDev );

        hipMemcpy( &t, tDev, size, hipMemcpyDeviceToHost );
        hipMemcpy( &V[0], VDev, nSize, hipMemcpyDeviceToHost );
        hipMemcpy( &mask[0], maskDev, bSize, hipMemcpyDeviceToHost );

        hipFree( dtDev );
        hipFree( IDev );
        hipFree( VPeakDev );
        hipFree( VResetDev );
        hipFree( paramSpecDev );
        hipFree( tDev );
        hipFree( VDev );
        hipFree( maskDev );

    }

}

